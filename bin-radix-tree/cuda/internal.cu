#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

struct Node
{
    bool isLeaf;
    int data;

    int left;
    bool leftIsLeaf;

    int right;
    bool rightIsLeaf;
};

__device__ int signDevice(int x)
{
    return (x >= 0) - (x < 0);
}

__device__ int commonPrefixLengthDevice(int a, int b)
{
    if (a == b)
        return sizeof(int) * 8; // If numbers are equal, full bit length match

    int xorVal = a ^ b;   // Find differing bits
    return __clz(xorVal); // Count leading zeros
}

__device__ int computeCPLDevice(int *randomInts, int size, int i, int j)
{
    if (j >= size || j < 0)
        return -1;

    return commonPrefixLengthDevice(randomInts[i], randomInts[j]);
}

__global__ void buildInternalKernel(int *randomInts, Node *leaves, Node *internalNodes, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size)
        return;

    // determine range direction
    int d = signDevice(computeCPLDevice(randomInts, size + 1, i, i + 1) - computeCPLDevice(randomInts, size + 1, i, i - 1));

    int minCPL = computeCPLDevice(randomInts, size + 1, i, i - d);

    // determine range extent
    int ub = i;
    while (computeCPLDevice(randomInts, size + 1, i, ub + d) > minCPL)
    {
        ub += d;
    }

    int dNode = computeCPLDevice(randomInts, size + 1, i, ub);

    // find split point
    int s = 0;
    while (computeCPLDevice(randomInts, size + 1, i, i + (s + 1) * d) > dNode)
    {
        s++;
    }

    int y = i + s * d + min(d, 0);

    int left = y;
    bool leftIsLeaf = false;

    int right = y + 1;
    bool rightIsLeaf = false;
    if (min(i, ub) == y)
        leftIsLeaf = true;

    if (max(i, ub) == y + 1)
        rightIsLeaf = true;

    internalNodes[i].data = i;
    internalNodes[i].left = left;
    internalNodes[i].leftIsLeaf = leftIsLeaf;

    internalNodes[i].right = right;
    internalNodes[i].rightIsLeaf = rightIsLeaf;
}

void buildInternalNodes(std::vector<int> &randomInts, std::vector<Node> &internalNodes, std::vector<Node> &leaves)
{
    int size = internalNodes.size();

    // Allocate memory on GPU
    int *d_randomInts;
    Node *d_internalNodes, *d_leaves;

    hipMalloc(&d_randomInts, randomInts.size() * sizeof(int));
    hipMalloc(&d_internalNodes, internalNodes.size() * sizeof(Node));
    hipMalloc(&d_leaves, leaves.size() * sizeof(Node));

    // Copy data from host to device
    hipMemcpy(d_randomInts, randomInts.data(), randomInts.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_internalNodes, internalNodes.data(), internalNodes.size() * sizeof(Node), hipMemcpyHostToDevice);
    hipMemcpy(d_leaves, leaves.data(), leaves.size() * sizeof(Node), hipMemcpyHostToDevice);

    // Configure kernel launch
    double threadsPerBlock = 256;
    int blocksPerGrid = ceil(size / threadsPerBlock);

    // Launch kernel
    buildInternalKernel<<<blocksPerGrid, threadsPerBlock>>>(d_randomInts, d_leaves, d_internalNodes, size);
    hipDeviceSynchronize(); // Wait for GPU execution to complete

    // Copy results back to host
    hipMemcpy(internalNodes.data(), d_internalNodes, internalNodes.size() * sizeof(Node), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_randomInts);
    hipFree(d_internalNodes);
    hipFree(d_leaves);
}